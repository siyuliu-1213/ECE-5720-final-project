/*
 * ECE 5720 Parallel Computing final project
 * Substring matching with CUDA
 * Shicong Li sl3295
 * Siyu Liu sl3282
 * Cornell University
 *
 * Compile : /usr/local/cuda-10.1/bin/nvcc -arch=compute_52 -o KMP_cuda KMP_cuda.cu
 * Run     : ./KMP_cuda
 */

#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <math.h>
#define n_s 1e9
#define n_p 4
#define M 10000
#define N 8
#define BILLION 1E9L

__global__ void match(char *dev_s, char *dev_p, int *dev_lps, uint *dev_res_map)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    // calculate the start point and end point
    int start = i * n_s / (M * N);
    int end = (i + 1) * n_s / (M * N) + n_p - 1;
    // local variable for KMP matching
    int id_p = 0;
    int id_s = start;
    while(id_s < end) {
        if(dev_s[id_s] == dev_p[id_p]) {
            id_s++;
            id_p++;
        }

        if(id_p == n_p) {
            int idx = id_s - id_p;
            dev_res_map[idx / 32] |= 1 << (idx % 32);
            id_p = dev_lps[id_p - 1];
        }

        else if(id_s < end && dev_s[id_s] != dev_p[id_p]) {
            if(id_p != 0) id_p = dev_lps[id_p - 1];
            else id_s++;
        }
    }
}

void computeLPS(char* p, int* lps, int n) {
    // Initialization of lps array
    int len = 0;
    lps[0] = 0;

    int id = 1;
    while(id < n) {
        // record and move forward the pointer if character are identical
        if(p[id] == p[len]) {
            len++;
            lps[id] = len;
            id++;
        }

        // If not, move the  id pointer backward and compare again
        else {
            if(len != 0) len = lps[len - 1];
            else {
                lps[id] = 0;
                id++;
            }
        }
    }
}

int main() {
    char *s, *p, *dev_s, *dev_p;
    int *lps, *dev_lps;
    uint *res_map, *dev_res_map;

    s = (char *) malloc((n_s + n_p - 1) * sizeof(char));
    p = (char *) malloc(n_p * sizeof(char));
    lps = (int *) malloc(n_p * sizeof(int));
    res_map = (uint *) calloc((n_s/32), sizeof(uint));
    FILE * fptr = fopen( "../data_5.txt" , "r");
    fgets(s, n_s + 1, fptr);
    fgets(p, n_p + 1, fptr);
    fclose(fptr);
    for(int i = n_s; i < n_s + n_p - 1; i++) s[i] = p[1] + 1;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMalloc( (void**)&dev_s, (n_s + n_p - 1)*sizeof(char)); 
    hipMalloc( (void**)&dev_p, n_p*sizeof(char));
    hipMalloc( (void**)&dev_lps, n_p*sizeof(int)); 
    hipMalloc( (void**)&dev_res_map, (n_s/32)*sizeof(uint)); 

    computeLPS(p, lps, n_p);
    hipMemcpy(dev_s, s, (n_s + n_p - 1)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_p, p, n_p*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_lps, lps, n_p*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_res_map, res_map, (n_s/32)*sizeof(uint), hipMemcpyHostToDevice);

    match<<<M, N>>>(dev_s, dev_p, dev_lps, dev_res_map);

    hipMemcpy(res_map, dev_res_map, (n_s/32)*sizeof(uint), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Total time is %lf\n", milliseconds);

    hipFree(dev_s); hipFree(dev_p); hipFree(dev_lps); hipFree(dev_res_map);
    free(s); free(p); free(lps); free(res_map);
}